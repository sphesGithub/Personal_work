#include "hip/hip_runtime.h"
#include <stdlib.h>
#include <stdio.h>
#include <math.h>
#include <hip/hip_runtime.h>
//NEW libraries to handle timing
#include <Winsock2.h>
#include <stdint.h> 
#include <Windows.h>


// size of plate
#define COLUMNS 1000
#define ROWS 1000

#ifndef MAX_ITER
#define MAX_ITER 100
#endif

// largest permitted change in temp (This value takes about 3400 steps)
#define MAX_TEMP_ERROR 0.01


int gettimeofday(struct timeval * tp, struct timezone * tzp)
{
	//NEW code taken from Stack Overflow to calculate timings on Windows
	static const uint64_t EPOCH = ((uint64_t)116444736000000000ULL);

	SYSTEMTIME  system_time;
	FILETIME    file_time;
	uint64_t    time;

	GetSystemTime(&system_time);
	SystemTimeToFileTime(&system_time, &file_time);
	time = ((uint64_t)file_time.dwLowDateTime);
	time += ((uint64_t)file_time.dwHighDateTime) << 32;

	tp->tv_sec = (long)((time - EPOCH) / 10000000L);
	tp->tv_usec = (long)(system_time.wMilliseconds * 1000);
	return 0;
}

// Kernel function to initialize the temperature array
__global__ void initialize(double *Temperature_last) {
    int i = blockIdx.y * blockDim.y + threadIdx.y;
    int j = blockIdx.x * blockDim.x + threadIdx.x;

    // Initialize the interior and boundary of the grid
    if (i <= ROWS + 1 && j <= COLUMNS + 1) {
        // Initialize interior to 0.0
        Temperature_last[i * (COLUMNS + 2) + j] = 0.0;

        // Set left and right boundaries
        if (j == 0) {
            Temperature_last[i * (COLUMNS + 2) + j] = 0.0;
        }
        if (j == COLUMNS + 1) {
            Temperature_last[i * (COLUMNS + 2) + j] = (100.0 / ROWS) * i;
        }

        // Set top and bottom boundaries
        if (i == 0) {
            Temperature_last[i * (COLUMNS + 2) + j] = 0.0;
        }
        if (i == ROWS + 1) {
            Temperature_last[i * (COLUMNS + 2) + j] = (100.0 / COLUMNS) * j;
        }
    }
}

__device__ double atomicMax(double* address, double val) {
    
    /**
     * I found this function one
     *reads the 32-bit or 64-bit word old located at the address address in global or shared memory, computes the maximum of old and val, 
     *and stores the result back to memory at the same address. 
     *These three operations are performed in one atomic transaction. The function returns old.
     *link:https://docs.nvidia.com/cuda/cuda-c-programming-guide/index.html?highlight=atomicMax#atomicmax
    */
    unsigned long long int* address_as_ull = (unsigned long long int*)address;
    unsigned long long int old = *address_as_ull, assumed;

    do {
        assumed = old;
        old = atomicCAS(address_as_ull, assumed,
                        __double_as_longlong(fmax(val, __longlong_as_double(assumed))));
    } while (assumed != old);

    return __longlong_as_double(old);
}
//Kernel function to calculate the max difference
// Computes the maximum temperature change (error) between the current and previous iterations, and updates the previous temperature array.
__global__ void max_dt(double *Temp, double *Temp_last, double *d_max_error) {
    int i = blockIdx.y * blockDim.y + threadIdx.y + 1;
    int j = blockIdx.x * blockDim.x + threadIdx.x + 1;

    double local_max_error = 0.0;

    if (i <= ROWS && j<=COLUMNS) {
        double diff = fabs(Temp[i * (COLUMNS + 2) + j] - Temp_last[i * (COLUMNS + 2) + j]);
        local_max_error = diff;
        Temp_last[i * (COLUMNS + 2) + j] = Temp[i * (COLUMNS + 2) + j];
    }

    // Use atomicMax to update the global maximum error
    atomicMax(d_max_error, local_max_error);
}

//kernerl function to calculate the laplace
__global__ void laplace(double *Temp,double *Temp_last)
{
    int i = blockIdx.y * blockDim.y + threadIdx.y;
    int j = blockIdx.x * blockDim.x + threadIdx.x;

        // main calculation: average my four neighbors
        if (i <= ROWS + 1 && j <= COLUMNS + 1)
        {
            Temp[i * (COLUMNS + 2) + j]=0.25 * (Temp_last[(i+1) * (COLUMNS + 2) + j] + Temp_last[(i-1) * (COLUMNS + 2) + j] +
                                            Temp_last[i * (COLUMNS + 2) + (j+1)] + Temp_last[i * (COLUMNS + 2) + (j-1)]);
        }
        
}

// print diagonal in bottom right corner where most action is

int main(int argc, char const *argv[])
{

    struct timeval start_time, stop_time;  		 // timers
    int max_iterations = MAX_ITER;

    dim3 blockDim(16, 16);                                                                               // Define thread block dimensions
    dim3 gridDim((COLUMNS + 2 + blockDim.x - 1) / blockDim.x, (ROWS + 2 + blockDim.y - 1) / blockDim.y); // Define grid dimensions

    // Allocate memory on the host and device for Temperature_last
    double *Temperature_last_host = (double *)malloc(sizeof(double) * (ROWS + 2) * (COLUMNS + 2));
    double *Temperature_last_dev;
    hipMalloc((void **)&Temperature_last_dev, sizeof(double) * (ROWS + 2) * (COLUMNS + 2));

    // Initialize Temperature_last on the device
    initialize<<<gridDim, blockDim>>>(Temperature_last_dev);
    hipDeviceSynchronize(); // Ensure kernel execution is complete

    // Copy Temperature_last from device to host
    hipMemcpy(Temperature_last_host, Temperature_last_dev, sizeof(double) * (ROWS + 2) * (COLUMNS + 2), hipMemcpyDeviceToHost);


    double *Temperature_host = (double *)malloc(sizeof(double) * (ROWS + 2) * (COLUMNS + 2));
    double *Temperature_dev;
    
    hipMalloc((void **)&Temperature_dev, sizeof(double) * (ROWS + 2) * (COLUMNS + 2));

    int iteration=1;                                     // current iteration
    double dt=100;  
    double *d_max_error;
    hipMalloc((void **)&d_max_error, sizeof(double));
    double max_error = 0.0;


    // do until error is minimal or until max steps
    while ( dt > MAX_TEMP_ERROR && iteration <= max_iterations ) {
    
        // Initialize Temperature_last on the device
        laplace<<<gridDim, blockDim>>>(Temperature_dev,Temperature_last_dev);
        hipDeviceSynchronize(); // Ensure kernel execution is complete
        dt = 0.0; // reset largest temperature change
        max_dt<<<gridDim, blockDim>>>(Temperature_dev, Temperature_last_dev, d_max_error);
        hipDeviceSynchronize();

        hipMemcpy(&max_error, d_max_error, sizeof(double), hipMemcpyDeviceToHost);
        dt = max_error;

	    iteration++;
    }
    

    // Copy Temperature_last from device to host
    hipMemcpy(Temperature_host, Temperature_dev, sizeof(double) * (ROWS + 2) * (COLUMNS + 2), hipMemcpyDeviceToHost);

    bool viewOriginal = true;// please change the value to false to see the full plate
    if(viewOriginal){
        int i;
    printf("---------- Iteration number: %d ------------\n", iteration-1);
    for(i = ROWS-5; i <= ROWS; i++) {
        printf("[%d,%d]: %5.2f  ", i, i, Temperature_host[i * (COLUMNS + 2) + i]);
    }
    printf("\n");

    }
    else{

    printf("Final temperature---------------------------------------------------\n\n");
     for (int i = ROWS-5; i <= ROWS ; i++)
    {
        for (int j = COLUMNS-5; j <= COLUMNS ; j++)
        {
            printf("[%d,%d]: %5.2f  ", i, j, Temperature_host[i * (COLUMNS + 2) + j]);
        }
        printf("\n");
    }

    }

     gettimeofday(&stop_time,NULL);
    float diff = fabs(( (stop_time.tv_sec-start_time.tv_sec)*1000000 + (stop_time.tv_usec - start_time.tv_usec) )/1000000.0);
        
    printf("\nMax error at iteration %d was %f\n", iteration-1, dt);
    printf("Total time was %f seconds\n", diff);
    
      // Free allocated memory
    free(Temperature_last_host);
    free(Temperature_host);
    hipFree(Temperature_last_dev);
    hipFree(Temperature_dev);
    hipFree(d_max_error);
    return 0;
}
